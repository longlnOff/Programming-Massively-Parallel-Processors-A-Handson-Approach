
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

__global__
void vecAddKernel(float* A, float* B, float* C, size_t N)
{
    size_t i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
    
}

void vecAdd(float* A, float* B, float* C, int N) {
    size_t size = N * sizeof(float);
    float* A_d, * B_d, * C_d;

    // Part 1: Allocate device memory for A, B and C
    // Copy A and B to device memory
    hipMalloc((void**)(&A_d), size);
    hipMalloc((void**)(&B_d), size);
    hipMalloc((void**)(&C_d), size);
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);


    // Part 2: Call kernel - to launch a grid of threads
    // to perform the actual vector addition
    vecAddKernel<<<ceil(N/256.0), 256>>>(A_d, B_d, C_d, N);


    // Part 3: Copy C from the device memory 
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
    // Free device vectors
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(void)
{
    size_t N = 1000;
    std:: cout << "Number of elements in array: ";
    std::cout << N
              << std::setw(8)
              << std::endl;
    size_t size = N * sizeof(float);

    float* A_h {(float*)malloc(size)};
    float* B_h {(float*)malloc(size)};
    float* C_h {(float*)malloc(size)};

    vecAdd(A_h, B_h, C_h, N);

    for (size_t i {0}; i < 10; ++i)
        std::cout << A_h[i] << std::endl;


    return 0;
}