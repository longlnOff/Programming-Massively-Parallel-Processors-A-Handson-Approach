
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

__global__
void vecAddKernel(float* A, float* B, float* C, int N)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
    
}

void vecAdd(float* A, float* B, float* C, int N) {
    int size = N * sizeof(float);
    float* A_d, * B_d, * C_d;

    // Part 1: Allocate device memory for A, B and C
    // Copy A and B to device memory
    hipMalloc((void**)(&A_d), size);
    hipMalloc((void**)(&B_d), size);
    hipMalloc((void**)(&C_d), size);
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);


    // Part 2: Call kernel - to launch a grid of threads
    // to perform the actual vector addition
    vecAddKernel<<<ceil(N/256.0), 256>>>(A_d, B_d, C_d, N);


    // Part 3: Copy C from the device memory 
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
    // Free device vectors
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(void)
{
    int N = 1000;
    std:: cout << "Number of elements in array: ";
    std::cout << N
              << std::endl;
    int size = N * sizeof(float);

    float* A_h {(float*)malloc(size)};
    float* B_h {(float*)malloc(size)};
    float* C_h {(float*)malloc(size)};

    vecAdd(A_h, B_h, C_h, N);

    for (int i = 0; i < N; i++) {
        A_h[i] = 1.0f;
        B_h[i] = 2.0f;
    }

    for (int i {0}; i < 10; ++i)
    {
        std::cout << "A: " << A_h[i];
        std::cout << " B: " << B_h[i];
        std::cout << " C: " << C_h[i] << std::endl;
    }


    return 0;
}