
#include <hip/hip_runtime.h>
#include <iostream>


void vecAdd(float* A_h, float* B_h, float* C_h, int N) {
    for (int i = 0; i < N; i++) {
        C_h[i] = A_h[i] + B_h[i];
    }
}

int main(void)
{
    int N = 1 << 20;
    size_t size = N * sizeof(float);

    float* A_h = (float*)malloc(size);
    float* B_h = (float*)malloc(size);
    float* C_h = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        A_h[i] = 1.0f;
        B_h[i] = 2.0f;
    }


    for (int i = 0; i < N; i++) 
    {
        A_h[i] = 1.0f;
        B_h[i] = 2.0f;
    }

    vecAdd(A_h, B_h, C_h, N);



    for (int i {0}; i < 10; ++i)
    {
        std::cout << "A: " << A_h[i];
        std::cout << " B: " << B_h[i];
        std::cout << " C: " << C_h[i] << std::endl;
    }



    free(A_h);
    free(B_h);
    free(C_h);

    return 0;
}