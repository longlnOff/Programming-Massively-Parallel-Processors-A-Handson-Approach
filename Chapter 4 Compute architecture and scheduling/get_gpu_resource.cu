#include "hip/hip_runtime.h"
#include <iostream>


int main(void)
{
    hipDeviceProp_t devProp {};
    int devCount {};
    hipGetDeviceCount(&devCount);

    for (int i {0}; i < devCount; ++i)
    {
        hipGetDeviceProperties(&devProp, i);
        std::cout << "Device " << i << ": " << devProp.name << std::endl;
        std::cout << "  Compute capability: " << devProp.major << "." << devProp.minor << std::endl;
        std::cout << "  Total global memory: " << devProp.totalGlobalMem << std::endl;
        std::cout << "  Total constant memory: " << devProp.totalConstMem << std::endl;
        std::cout << "  Shared memory per block: " << devProp.sharedMemPerBlock << std::endl;
        std::cout << "  Registers per block: " << devProp.regsPerBlock << std::endl;
        std::cout << "  Warp size: " << devProp.warpSize << std::endl;
        std::cout << "  Max threads per block: " << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Max threads dimensions: " << devProp.maxThreadsDim[0] << " x " << devProp.maxThreadsDim[1] << " x " << devProp.maxThreadsDim[2] << std::endl;
        std::cout << "  Max grid size: " << devProp.maxGridSize[0] << " x " << devProp.maxGridSize[1] << " x " << devProp.maxGridSize[2] << std::endl;
        std::cout << "  Clock rate: " << devProp.clockRate << std::endl;
        std::cout << "  Memory clock rate: " << devProp.memoryClockRate << std::endl;
        std::cout << "  Memory bus width: " << devProp.memoryBusWidth << std::endl;
        std::cout << "  Peak memory bandwidth: " << 2.0 * devProp.memoryClockRate * (devProp.memoryBusWidth / 8) / 1.0e6 << " GB/s" << std::endl;
        std::cout << "  L2 cache size: " << devProp.l2CacheSize << std::endl;
        std::cout << "  Multi-processor count: " << devProp.multiProcessorCount << std::endl;

        return 0;
    }
}