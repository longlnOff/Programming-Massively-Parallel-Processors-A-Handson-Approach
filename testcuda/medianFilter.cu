#include "hip/hip_runtime.h"
#include "medianFilter.h"
__global__ void kernelMedianFilter(uchar3* d_inputMat, uchar* d_kernelMat)
{
    return;
}
void cudaMedianCaller(const cv::Mat& inputMat, cv::Mat& kernelMat)
{
    // allocate device pointers
    uchar3 *d_inputMat;
    uchar  *d_kernelMat;
    hipMalloc(&d_inputMat,  inputMat.total() * sizeof(uchar3));
    hipMalloc(&d_kernelMat, kernelMat.total() * sizeof(uchar));

    // copy from host to device
    hipMemcpy(d_inputMat, inputMat.ptr<uchar3>(0), inputMat.total() * sizeof(uchar3), hipMemcpyHostToDevice);
    hipMemcpy(d_kernelMat, kernelMat.ptr<uchar>(0), kernelMat.total() * sizeof(uchar), hipMemcpyHostToDevice);

    // call CUDA kernel
    kernelMedianFilter <<<1, 1>>> (d_inputMat, d_kernelMat);

    // free
    hipFree(d_inputMat);
    hipFree(d_kernelMat);
}