#include "hip/hip_runtime.h"
#include "MatrixMultiplication.h"


// P = M dot N
__global__
void MatrixMultiplicationRowKernel(int *M, int *N, int *P, const uint MatrixSize)
{
    uint row_index {blockDim.y * blockIdx.y + threadIdx.y};
    if (row_index < MatrixSize)
    {
        for (uint column_index {0}; column_index < MatrixSize; ++column_index)
        {
            int value {0};
            uint P_index {row_index * MatrixSize + column_index};
            // P[s,s] = M[s,s]*N[s,s]
            for (uint index {0}; index < MatrixSize; ++index)
            {
                uint M_index {row_index * MatrixSize + index};
                uint N_index {index * MatrixSize + column_index};
                value += M[M_index] * N[N_index];
            }
            P[P_index] = value;
        }
    }

}


void MatrixMultiplicationRow(int *M_h, int *N_h, int *P_h, const size_t MatrixSize)
{
    size_t SizeAlloc {MatrixSize * MatrixSize * sizeof(int)};
    int *M_d {nullptr};
    int *N_d {nullptr};
    int *P_d {nullptr};

    // Allocate device's memory for Pin and Pout
    assert(hipMalloc((void **) &M_d, SizeAlloc) == hipSuccess);
    assert(hipMalloc((void **) &N_d, SizeAlloc) == hipSuccess);
    assert(hipMalloc((void **) &P_d, SizeAlloc) == hipSuccess);

    // Copy Pin to device memory
    assert(hipMemcpy(M_d, M_h, SizeAlloc, hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(N_d, N_h, SizeAlloc, hipMemcpyHostToDevice) == hipSuccess);

    // // Call kernel function
    dim3 Blockdim {static_cast<uint>(16), 
                    static_cast<uint>(16), 
                    static_cast<uint>(1)};
    dim3 Griddim {static_cast<uint>(ceil(MatrixSize/16.0)), 
                   static_cast<uint>(ceil(MatrixSize/16.0)), 
                   static_cast<uint>(1)};

    MatrixMultiplicationRowKernel<<<Griddim, Blockdim>>>(M_d, N_d, P_d, MatrixSize);

    // Copy Pout from device to host
    assert(hipMemcpy(P_h, P_d, SizeAlloc, hipMemcpyDeviceToHost) == hipSuccess);
    // Free device's memory
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
}
