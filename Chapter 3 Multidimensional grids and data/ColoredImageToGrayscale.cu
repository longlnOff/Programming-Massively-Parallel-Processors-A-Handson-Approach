
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

// Convert colored image to grayscale kernel
__global__
void Colored2GrayscaleKernel(unsigned char* Pin, unsigned char* Pout, int width, int height)
{
    size_t row_index {blockDim.y * blockIdx.y + threadIdx.y};
    size_t column_index {blockDim.x * blockIdx.x + threadIdx.x};

    // check whether thread grid fit with images
    if (column_index < width && row_index < height)
    {
        // Get grayscale offset
        size_t GrayOffset {row_index * width + column_index};

        // Get colored offset
        size_t ColoredOffset {GrayOffset * 3};

        // Get RGB values
        unsigned char RedValue      = Pin[ColoredOffset + 0];
        unsigned char GreenValue    = Pin[ColoredOffset + 1];
        unsigned char BlueValue     = Pin[ColoredOffset + 2];

        // Get gray value from RGB
        Pout[GrayOffset] = 0.21f*RedValue + 0.71f*GreenValue + 0.07f*BlueValue;
    }

}

void Colored2Grayscale(unsigned char* Pin_h, unsigned char* Pout_h, int width, int height)
{
    size_t SizeAllocColored {static_cast<size_t>(3 * width * height * sizeof(unsigned char))};
    size_t SizeAllocGray {static_cast<size_t>(width * height * sizeof(unsigned char))};
    
    unsigned char* Pin_d {};
    unsigned char* Pout_d {};

    // Allocate device's memory for Pin and Pout
    assert(hipMalloc((void **) &Pin_d, SizeAllocColored) == hipSuccess);
    assert(hipMalloc((void **) &Pout_d, SizeAllocGray) == hipSuccess);

    // Copy Pin to device memory
    assert(hipMemcpy(Pin_d, Pin_h, SizeAllocColored, hipMemcpyHostToDevice) == hipSuccess);

    // Call kernel function
    dim3 Blockdim {static_cast<unsigned int>(16), 
                    static_cast<unsigned int>(16), 
                    static_cast<unsigned int>(1)};
                    
    dim3 Griddim {static_cast<unsigned int>(ceil(width/16.0)), 
                   static_cast<unsigned int>(ceil(height/16.0)), 
                   static_cast<unsigned int>(1)};

    Colored2GrayscaleKernel<<<Griddim, Blockdim>>>(Pin_d, Pout_d, width, height);

    std::cout << "Here" << std::endl;

    // Copy Pout from device to host
    assert(hipMemcpy(Pout_h, Pout_d, SizeAllocGray, hipMemcpyDeviceToHost) == hipSuccess);

    // Free device's memory
    hipFree(Pin_d);
    hipFree(Pout_d);

}


int main()
{
    // int width {}, height {}, componentCount {};
    // unsigned char* ImageData {stbi_load("1.png", &width, &height, &componentCount, 3)};
    // std::cout << "Channel in file: " << componentCount << std::endl;

    // // Validate image size
    // std::cout << width << " X " << height << std::endl;

    // unsigned char* GrayImage {};

    // Colored2Grayscale(ImageData, GrayImage, width, height);
    // std::cout << "Hello" << std::endl;
    // // Write image back to disk
    // // stbi_write_png("image_gray.png", width, height, 3, GrayImage, 3 * width);

    return 0;
}