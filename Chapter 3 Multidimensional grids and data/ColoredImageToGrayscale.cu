#include "hip/hip_runtime.h"
#include "ColoredImageToGrayscale.h"



// Convert colored image to grayscale kernel
// __global__
// void Colored2GrayscaleKernel(unsigned char* Pin, unsigned char* Pout, int width, int height)
// {
//     size_t row_index {blockDim.y * blockIdx.y + threadIdx.y};
//     size_t column_index {blockDim.x * blockIdx.x + threadIdx.x};

//     // check whether thread grid fit with images
//     if (column_index < width && row_index < height)
//     {
//         // Get grayscale offset
//         size_t GrayOffset {row_index * width + column_index};

//         // Get colored offset
//         size_t ColoredOffset {GrayOffset * 3};

//         // Get RGB values
//         unsigned char RedValue      = Pin[ColoredOffset + 0];
//         unsigned char GreenValue    = Pin[ColoredOffset + 1];
//         unsigned char BlueValue     = Pin[ColoredOffset + 2];

//         // Get gray value from RGB
//         Pout[GrayOffset] = 0.21f*RedValue + 0.71f*GreenValue + 0.07f*BlueValue;
//     }

// }

// void Colored2Grayscale(unsigned char* Pin_h, unsigned char* Pout_h, int width, int height)
// {

//     size_t SizeAllocColored {static_cast<size_t>(3 * width * height * sizeof(unsigned char))};
//     size_t SizeAllocGray {static_cast<size_t>(width * height * sizeof(unsigned char))};
    
//     unsigned char* Pin_d {};
//     unsigned char* Pout_d {};

//     // Allocate device's memory for Pin and Pout
//     assert(hipMalloc((void **) &Pin_d, SizeAllocColored) == hipSuccess);
//     assert(hipMalloc((void **) &Pout_d, SizeAllocGray) == hipSuccess);

//     // Copy Pin to device memory
//     assert(hipMemcpy(Pin_d, Pin_h, SizeAllocColored, hipMemcpyHostToDevice) == hipSuccess);

//     // Call kernel function
//     dim3 Blockdim {static_cast<unsigned int>(16), 
//                     static_cast<unsigned int>(16), 
//                     static_cast<unsigned int>(1)};
                    
//     dim3 Griddim {static_cast<unsigned int>(ceil(width/16.0)), 
//                    static_cast<unsigned int>(ceil(height/16.0)), 
//                    static_cast<unsigned int>(1)};

//     Colored2GrayscaleKernel<<<Griddim, Blockdim>>>(Pin_d, Pout_d, width, height);

//     // Copy Pout from device to host
//     assert(hipMemcpy(Pout_h, Pout_d, SizeAllocGray, hipMemcpyDeviceToHost) == hipSuccess);
//     std::cout << "Colored image to grayscale image is done!" << std::endl;
//     // Free device's memory
//     hipFree(Pin_d);
//     hipFree(Pout_d);

// }


__host__
__device__
__attribute__((always_inline))
inline void color_to_grayscale_unit (u_char *input, u_char *output, const int width, const int height, int row, int col){
	int gray_offset = row*width + col;
	int rgb_offset = gray_offset * 3;
	output[gray_offset] = 0.07*input[rgb_offset + 2] + 0.71*input[rgb_offset + 1] + 0.21*input[rgb_offset + 0];
}

__global__
void color_to_grayscale_kernel(u_char *input, u_char *output, const int width, const int height){
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	if(col < width && row < height){
		color_to_grayscale_unit(input, output, width, height, row, col);
	}
}


void ch3__color_to_grayscale_device(u_char *h_input, u_char *h_output, const int width, const int height){
	u_char *d_input, *d_output;
	const int length = width*height;

	hipMalloc(&d_input, 3*length*sizeof(u_char));
	hipMalloc(&d_output, length*sizeof(u_char));

	hipMemcpy(d_input, h_input, 3*length*sizeof(u_char), hipMemcpyHostToDevice);

	dim3 block_dim(16, 16, 1);
	dim3 grid_dim(ceil(width/(double)16), ceil(height/(double)16), 1);

	color_to_grayscale_kernel<<<grid_dim, block_dim>>>(d_input, d_output, width, height);

	hipMemcpy(h_output, d_output, length*sizeof(u_char), hipMemcpyDeviceToHost);
    std::cout << "host output: " << int(h_output[0]) << std::endl;
    std::cout << "host input: " << int(h_input[0]) << std::endl;

	hipFree(d_input);
	hipFree(d_output);
}