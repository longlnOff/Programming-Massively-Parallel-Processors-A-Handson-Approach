#include "hip/hip_runtime.h"
#include <iostream>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "header_stb.h"
#include "stb_image_write.h"

// Convert colored image to grayscale kernel
__global__
void Colored2GrayscaleKernel(unsigned char* Pin, unsigned char* Pout, int width, int height)
{
    size_t row_index {blockDim.y * blockIdx.y + threadIdx.y};
    size_t column_index {blockDim.x * blockIdx.x + threadIdx.x};

    // check whether thread grid fit with images
    if (row_index < height && column_index < width)
    {
        // Get grayscale offset
        size_t GrayOffset {row_index * width + column_index};

        // Get colored offset
        size_t ColoredOffset {GrayOffset * 3};

        // Get RGB values
        unsigned char RedValue = Pin[ColoredOffset];
        unsigned char GreenValue = Pin[ColoredOffset + 1];
        unsigned char BlueValue = Pin[ColoredOffset + 2];

        // Get gray value from RGB
        Pout[GrayOffset] = 0.21f*RedValue + 0.71f*GreenValue + 0.07f*BlueValue;
    }

}

void Colored2Grayscale(unsigned char* Pin_h, unsigned char* Pout_h, int width, int height)
{
    size_t SizeAlloc {static_cast<size_t>(width * height * 3 * sizeof(unsigned char))};
    unsigned char* Pin_d {};
    unsigned char* Pout_d {};

    // Allocate device's memory for Pin and Pout
    hipMalloc((void **) &Pin_d, SizeAlloc);
    hipMalloc((void **) &Pout_d, SizeAlloc / 3);

    // Copy Pin to device memory
    hipMemcpy(Pin_d, Pin_h, SizeAlloc, hipMemcpyHostToDevice);

    // Call kernel function
    dim3 Blockdim {static_cast<unsigned int>(16), 
                    static_cast<unsigned int>(16), 
                    static_cast<unsigned int>(1)};
                    
    dim3 Griddim {static_cast<unsigned int>(ceil(width/16.0)), 
                   static_cast<unsigned int>(ceil(height/16.0)), 
                   static_cast<unsigned int>(1)};

    Colored2GrayscaleKernel<<<Griddim, Blockdim>>>(Pin_d, Pout_d, width, height);

    std::cout << "Here" << std::endl;

    // Copy Pout from device to host
    hipMemcpy(Pout_h, Pout_d, SizeAlloc / 3, hipMemcpyDeviceToHost);
    std::cout << Pout_h[0] << std::endl;

    // Free device's memory
    hipFree(Pin_d);
    hipFree(Pout_d);

}


int main()
{
    int width {}, height {}, componentCount {};
    unsigned char* ImageData {stbi_load("1.png", &width, &height, &componentCount, 3)};
    std::cout << sizeof(unsigned char) << std::endl;
    std::cout << "Channel in file: " << componentCount << std::endl;

    if (!ImageData)
    {
        std::cout << "Open PNG file." << std::endl;
    }


    // Validate image size
    std::cout << width << " X " << height << std::endl;

    unsigned char* GrayImage {};

    Colored2Grayscale(ImageData, GrayImage, width, height);
    // Write image back to disk
    stbi_write_png("image_gray.png", width, height, 3, GrayImage, 3 * width);


    return 0;
}