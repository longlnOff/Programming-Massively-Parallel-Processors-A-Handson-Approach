#include "hip/hip_runtime.h"
#include "BlurImage.h"


// Convert colored image to grayscale kernel
__global__
void BlurImageKernel(u_char* Pin, u_char* Pout, const int width, const int height, const int BLURSIZE)
{
    int row_index {static_cast<int>(blockDim.y * blockIdx.y + threadIdx.y)};
    int column_index {static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x)};

    // check whether thread grid fit with images
    if (column_index < width && row_index < height)
    {
        int PixVal = 0;
        int NumberOfPixels = 0;

        for (int blur_row {-BLURSIZE}; blur_row < BLURSIZE + 1; ++blur_row)
        {
            for (int blur_col {-BLURSIZE}; blur_col < BLURSIZE + 1; ++blur_col)
            {
                int BlurRow {row_index + blur_row};
                int BlurCol {column_index + blur_col};

                if (BlurRow >= 0 && BlurRow < height && BlurCol >= 0 && BlurCol < width)
                {
                    PixVal += Pin[BlurRow*width + BlurCol];
                    NumberOfPixels++;
                }
            }
        }
        
        Pout[row_index*width + column_index] = static_cast<u_char>(PixVal / NumberOfPixels);
    }

}

void BlurImage(u_char* Pin_h, u_char* Pout_h, const int width, const int height, const int BLURSIZE)
{

    const int SizeAlloc {(width * height * static_cast<int>(sizeof(u_char)))};
    
    u_char* Pin_d {nullptr};
    u_char* Pout_d {nullptr};

    // Allocate device's memory for Pin and Pout
    assert(hipMalloc((void **) &Pin_d, SizeAlloc) == hipSuccess);
    assert(hipMalloc((void **) &Pout_d, SizeAlloc) == hipSuccess);

    // Copy Pin to device memory
    assert(hipMemcpy(Pin_d, Pin_h, SizeAlloc, hipMemcpyHostToDevice) == hipSuccess);

    // Call kernel function
    dim3 Blockdim {static_cast<unsigned int>(16), 
                    static_cast<unsigned int>(16), 
                    static_cast<unsigned int>(1)};
    dim3 Griddim {static_cast<unsigned int>(ceil(width/16.0)), 
                   static_cast<unsigned int>(ceil(height/16.0)), 
                   static_cast<unsigned int>(1)};
    BlurImageKernel<<<Griddim, Blockdim>>>(Pin_d, Pout_d, width, height, BLURSIZE);

    // Copy Pout from device to host
    assert(hipMemcpy(Pout_h, Pout_d, SizeAlloc, hipMemcpyDeviceToHost) == hipSuccess);
    // Free device's memory
    hipFree(Pin_d);
    hipFree(Pout_d);
}




void ch3__blur_device(u_char *h_input, u_char *h_output, const int blur_size, const int width, const int height)
{
	u_char *d_input, *d_output;
	const int length = width*height;

	hipMalloc(&d_input, length*sizeof(u_char));
	hipMalloc(&d_output, length*sizeof(u_char));

	hipMemcpy(d_input, h_input, length*sizeof(u_char), hipMemcpyHostToDevice);



    dim3 block_dim {static_cast<unsigned int>(16), 
                    static_cast<unsigned int>(16), 
                    static_cast<unsigned int>(1)};
                    
    dim3 grid_dim {static_cast<unsigned int>(ceil(width/16.0)), 
                   static_cast<unsigned int>(ceil(height/16.0)), 
                   static_cast<unsigned int>(1)};


    BlurImageKernel<<<grid_dim, block_dim>>>(d_input, d_output, width, height, blur_size);

	hipMemcpy(h_output, d_output, length*sizeof(u_char), hipMemcpyDeviceToHost);

	hipFree(d_input);
	hipFree(d_output);
}